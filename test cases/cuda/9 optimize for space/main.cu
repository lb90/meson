#include <hip/hip_runtime.h>
#include <iostream>

int cuda_devices() {
    int result = 0;
    hipGetDeviceCount(&result);
    return result;
}


int main() {
    int n = cuda_devices();
    if (n == 0) {
        std::cout << "No Cuda hardware found. Exiting.\n";
        return 0;
    }

    std::cout << "Found " << n << "Cuda devices.\n";
    return 0;
}
